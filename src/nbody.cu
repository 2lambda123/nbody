#include "hip/hip_runtime.h"
/*
 *
 * nbody.cu
 *
 * N-body example that illustrates gravitational simulation.
 * This is the type of computation that GPUs excel at:
 * parallelizable, with lots of FLOPS per unit of external
 * memory bandwidth required.
 *
 * Requires: No minimum SM requirement.  If SM 3.x is not available,
 * this application quietly replaces the shuffle and fast-atomic
 * implementations with the shared memory implementation.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in
 *    the documentation and/or other materials provided with the
 *    distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>

#ifdef _OPENMP
#include <omp.h>
#endif

#ifdef _WIN32
#include <conio.h>
#else

#include <termios.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/select.h>

#include <math.h>

#include "libtime.h"

#include "chThread.h"
#include "chError.h"

#include "nbody.h"

enum nbodyAlgorithm_enum {
    CPU_AOS = 0,    /* This is the golden implementation */
    CPU_AOS_tiled,
    CPU_SOA,
    CPU_SOA_tiled,
#ifdef HAVE_SIMD
    CPU_SIMD,
#endif
    GPU_AOS,
    GPU_Shared,
    GPU_Const,
    multiGPU,
// SM 3.0 only
    GPU_Shuffle,
    GPU_AOS_tiled,
    GPU_AOS_tiled_const,
//    GPU_Atomic
};

static const char *rgszAlgorithmNames[] = {
    "CPU_AOS",
    "CPU_AOS_tiled",
    "CPU_SOA",
    "CPU_SOA_tiled",
#ifdef HAVE_SIMD
    "CPU_SIMD",
#endif
    "GPU_AOS",
    "GPU_Shared",
    "GPU_Const",
    "multiGPU",
// SM 3.0 only
    "GPU_Shuffle",
    "GPU_AOS_tiled",
    "GPU_AOS_tiled_const",
//    "GPU_Atomic"
};

static int kbhit(void)
{
    struct termios oldt, newt;
    int ch;
    int oldf;

    tcgetattr(STDIN_FILENO, &oldt);
    newt = oldt;
    newt.c_lflag &= ~(ICANON | ECHO);
    tcsetattr(STDIN_FILENO, TCSANOW, &newt);
    oldf = fcntl(STDIN_FILENO, F_GETFL, 0);
    fcntl(STDIN_FILENO, F_SETFL, oldf | O_NONBLOCK);

    ch = getchar();

    tcsetattr(STDIN_FILENO, TCSANOW, &oldt);
    fcntl(STDIN_FILENO, F_SETFL, oldf);

    if(ch != EOF)
    {
        ungetc(ch, stdin);
        return 1;
    }

    return 0;
}

// we only call getch() when kbhit() has told us there
// is a pending keystroke
static int
getch(void)
{
    return getchar();
}

#endif

static inline void
randomVector( float v[3] )
{
    float lenSqr;
    do {
        v[0] = rand() / (float) RAND_MAX * 2 - 1;
        v[1] = rand() / (float) RAND_MAX * 2 - 1;
        v[2] = rand() / (float) RAND_MAX * 2 - 1;
        lenSqr = v[0]*v[0]+v[1]*v[1]+v[2]*v[2];
    } while ( lenSqr > 1.0f );
}

static void
randomUnitBodies( float *pos, float *vel, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        randomVector( &pos[4*i] );
        randomVector( &vel[4*i] );
        pos[4*i+3] = 1.0f;  // unit mass
        vel[4*i+3] = 1.0f;
    }
}

static float
relError( float a, float b )
{
    if ( a == b ) return 0.0f;
    return fabsf(a-b)/b;
}

static int g_bCUDAPresent;
static int g_bSM30Present;

float *g_hostAOS_PosMass;
float *g_hostAOS_VelInvMass;
float *g_hostAOS_Force;

#ifndef NO_CUDA
static float *g_dptrAOS_PosMass;
static float *g_dptrAOS_Force;
#endif

// Buffer to hold the golden version of the forces, used for comparison
// Along with timing results, we report the maximum relative error with
// respect to this array.
static float *g_hostAOS_Force_Golden;

float *g_hostSOA_Pos[3];
float *g_hostSOA_Force[3];
float *g_hostSOA_Mass;
float *g_hostSOA_InvMass;

static size_t g_N;

static float g_softening = 0.1f;
static float g_damping = 0.995f;
static float g_dt = 0.016f;

#include "nbody_CPU_AOS.h"
#include "nbody_CPU_AOS_tiled.h"
#include "nbody_CPU_SOA.h"
#include "nbody_CPU_SOA_tiled.h"
#include "nbody_CPU_SIMD.h"

#ifndef NO_CUDA
#include "bodybodyInteraction.cuh"
#include "nbody_GPU_AOS.cuh"
#include "nbody_GPU_AOS_const.cuh"
#include "nbody_GPU_AOS_tiled.cuh"
#include "nbody_GPU_AOS_tiled_const.cuh"
//#include "nbody_GPU_SOA_tiled.cuh"
#include "nbody_GPU_Shuffle.cuh"
#include "nbody_GPU_Atomic.cuh"
#endif

static void
integrateGravitation_AOS( float *ppos, float *pvel, float *pforce, float dt, float damping, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        const int index = 4*i;
        const int indexForce = 3*i;

        float pos[3], vel[3], force[3];
        pos[0] = ppos[index+0];
        pos[1] = ppos[index+1];
        pos[2] = ppos[index+2];
        float invMass = pvel[index+3];

        vel[0] = pvel[index+0];
        vel[1] = pvel[index+1];
        vel[2] = pvel[index+2];

        force[0] = pforce[indexForce+0];
        force[1] = pforce[indexForce+1];
        force[2] = pforce[indexForce+2];

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        vel[0] += (force[0] * invMass) * dt;
        vel[1] += (force[1] * invMass) * dt;
        vel[2] += (force[2] * invMass) * dt;

        vel[0] *= damping;
        vel[1] *= damping;
        vel[2] *= damping;

        // new position = old position + velocity * deltaTime
        pos[0] += vel[0] * dt;
        pos[1] += vel[1] * dt;
        pos[2] += vel[2] * dt;

        ppos[index+0] = pos[0];
        ppos[index+1] = pos[1];
        ppos[index+2] = pos[2];

        pvel[index+0] = vel[0];
        pvel[index+1] = vel[1];
        pvel[index+2] = vel[2];
    }
}

static enum nbodyAlgorithm_enum g_Algorithm;

//
// g_maxAlgorithm is used to determine when to rotate g_Algorithm back to CPU_AOS
// If CUDA is present, it depends on SM version
//
// The shuffle and tiled implementations are SM 3.0 only.
//
// The CPU and GPU algorithms must be contiguous, and the logic in main() to
// initialize this value must be modified if any new algorithms are added.
//
static enum nbodyAlgorithm_enum g_maxAlgorithm;
static int g_bCrossCheck = 1;
static int g_bUseSIMDForCrossCheck = 0;
static int g_bNoCPU = 0;

static int
ComputeGravitation(
    float *ms,
    float *maxRelError,
    enum nbodyAlgorithm_enum algorithm,
    int bCrossCheck )
{
#ifndef NO_CUDA
    hipError_t status;
#endif
    int bSOA = 0;

    // AOS -> SOA data structures in case we are measuring SOA performance
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Pos[0][i]  = g_hostAOS_PosMass[4*i+0];
        g_hostSOA_Pos[1][i]  = g_hostAOS_PosMass[4*i+1];
        g_hostSOA_Pos[2][i]  = g_hostAOS_PosMass[4*i+2];
        g_hostSOA_Mass[i]    = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    if ( bCrossCheck ) {
#ifdef HAVE_SIMD
        if ( g_bUseSIMDForCrossCheck ) {
            ComputeGravitation_SIMD(
                            g_hostSOA_Force,
                            g_hostSOA_Pos,
                            g_hostSOA_Mass,
                            g_softening*g_softening,
                            g_N );
        } else
#endif
        {
            ComputeGravitation_SOA(
                            g_hostSOA_Force,
                            g_hostSOA_Pos,
                            g_hostSOA_Mass,
                            g_softening*g_softening,
                            g_N );
        }
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force_Golden[3*i+0] = g_hostSOA_Force[0][i];
            g_hostAOS_Force_Golden[3*i+1] = g_hostSOA_Force[1][i];
            g_hostAOS_Force_Golden[3*i+2] = g_hostSOA_Force[2][i];
        }
    }

    /* Reset the force values so we know the function tested did work. */
    memset(g_hostAOS_Force,    0, g_N * sizeof(float) * 3);
    memset(g_hostSOA_Force[0], 0, g_N * sizeof(float));
    memset(g_hostSOA_Force[1], 0, g_N * sizeof(float));
    memset(g_hostSOA_Force[2], 0, g_N * sizeof(float));

#ifndef NO_CUDA
    // CPU->GPU copies in case we are measuring GPU performance
    if ( g_bCUDAPresent ) {
        CUDART_CHECK( hipMemcpyAsync(
            g_dptrAOS_PosMass,
            g_hostAOS_PosMass,
            4*g_N*sizeof(float),
            hipMemcpyHostToDevice ) );
    }
#endif

    switch ( algorithm ) {
        case CPU_AOS:
            *ms = ComputeGravitation_AOS(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
        case CPU_AOS_tiled:
            *ms = ComputeGravitation_AOS_tiled(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
        case CPU_SOA:
            *ms = ComputeGravitation_SOA(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = 1;
            break;
        case CPU_SOA_tiled:
            *ms = ComputeGravitation_SOA_tiled(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = 1;
            break;
#ifdef HAVE_SIMD
        case CPU_SIMD:
            *ms = ComputeGravitation_SIMD(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = 1;
            break;
#endif
#ifndef NO_CUDA
        case GPU_AOS:
            *ms = ComputeGravitation_GPU_AOS(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_AOS_tiled:
            *ms = ComputeGravitation_GPU_AOS_tiled(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_AOS_tiled_const:
            *ms = ComputeGravitation_GPU_AOS_tiled_const(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
#if 0
// commented out - too slow even on SM 3.0
        case GPU_Atomic:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Atomic(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
#endif
        case GPU_Shared:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Shared(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Const:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeNBodyGravitation_GPU_AOS_const(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case GPU_Shuffle:
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 3*g_N*sizeof(float) ) );
            *ms = ComputeGravitation_GPU_Shuffle(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 3*g_N*sizeof(float), hipMemcpyDeviceToHost ) );
            break;
        case multiGPU:
            memset( g_hostAOS_Force, 0, 3*g_N*sizeof(float) );
            *ms = ComputeGravitation_multiGPU(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
#endif
        default:
            fprintf(stderr, "Unrecognized algorithm index: %d\n", algorithm);
            abort();
    }

    // SOA -> AOS
    if ( bSOA ) {
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force[3*i+0] = g_hostSOA_Force[0][i];
            g_hostAOS_Force[3*i+1] = g_hostSOA_Force[1][i];
            g_hostAOS_Force[3*i+2] = g_hostSOA_Force[2][i];
        }
    }

    *maxRelError = 0.0f;
    if ( bCrossCheck ) {
        float max = 0.0f;
        for ( size_t i = 0; i < 3*g_N; i++ ) {
            float err = relError( g_hostAOS_Force[i], g_hostAOS_Force_Golden[i] );
            if ( err > max ) {
                max = err;
            }
        }
        *maxRelError = max;
    }

    integrateGravitation_AOS(
        g_hostAOS_PosMass,
        g_hostAOS_VelInvMass,
        g_hostAOS_Force,
        g_dt,
        g_damping,
        g_N );
    return 0;
#ifndef NO_CUDA
Error:
    return 1;
#endif
}

static worker_thread_t *g_GPUThreadPool;
int g_numGPUs;

struct gpuInit_struct
{
    int iGPU;

    hipError_t status;
};

static int initializeGPU( void *_p )
{
    hipError_t status;

    struct gpuInit_struct *p = (struct gpuInit_struct *) _p;
    CUDART_CHECK( hipSetDevice( p->iGPU ) );
    CUDART_CHECK( hipFree(0) );
Error:
    p->status = status;
    return 0;
}

static int teardownGPU( void *_p )
{
    hipError_t status;

    struct gpuInit_struct *p = (struct gpuInit_struct *) _p;
    CUDART_CHECK( hipSetDevice( p->iGPU ) );
    CUDART_CHECK( hipDeviceReset() );
Error:
    p->status = status;
    return 0;
}

static int
freeArrays(void)
{
#ifndef NO_CUDA
    hipError_t status;

    if ( g_bCUDAPresent ) {
        CUDART_CHECK( hipDeviceSynchronize() );
        CUDART_CHECK( hipHostFree( g_hostAOS_PosMass ) );
        for ( size_t i = 0; i < 3; i++ ) {
            CUDART_CHECK( hipHostFree( g_hostSOA_Pos[i] ) );
            CUDART_CHECK( hipHostFree( g_hostSOA_Force[i] ) );
        }
        CUDART_CHECK( hipHostFree( g_hostAOS_Force ) );
        CUDART_CHECK( hipHostFree( g_hostAOS_Force_Golden ) );
        CUDART_CHECK( hipHostFree( g_hostAOS_VelInvMass ) );
        CUDART_CHECK( hipHostFree( g_hostSOA_Mass ) );
        CUDART_CHECK( hipHostFree( g_hostSOA_InvMass ) );

        CUDART_CHECK( hipFree( g_dptrAOS_PosMass ) );
        CUDART_CHECK( hipFree( g_dptrAOS_Force ) );
    } else
#endif
    {
        free(g_hostAOS_PosMass);
        for ( size_t i = 0; i < 3; i++ ) {
            free(g_hostSOA_Pos[i]);
            free(g_hostSOA_Force[i]);
        }
        free(g_hostAOS_Force);
        free(g_hostAOS_Force_Golden);
        free(g_hostAOS_VelInvMass);
        free(g_hostSOA_Mass);
        free(g_hostSOA_InvMass);
    }
    return 0;
#ifndef NO_CUDA
Error:
    fprintf(stderr, "Failed to clean up memory.\n");
    return 1;
#endif
}

static int
allocArrays(void)
{
#ifndef NO_CUDA
    hipError_t status;

    if ( g_bCUDAPresent ) {
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_PosMass, 4*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        for ( size_t i = 0; i < 3; i++ ) {
            CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Pos[i], g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
            CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Force[i], g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        }
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force, 3*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force_Golden, 3*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_VelInvMass, 4*g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Mass, g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_InvMass, g_N*sizeof(float), hipHostMallocPortable|hipHostMallocMapped ) );

        CUDART_CHECK( hipMalloc( &g_dptrAOS_PosMass, 4*g_N*sizeof(float) ) );
        CUDART_CHECK( hipMalloc( (void **) &g_dptrAOS_Force, 3*g_N*sizeof(float) ) );
    } else
#endif
    {
        g_hostAOS_PosMass = (float *)valloc(sizeof(float) * 4 * g_N);
        if (!g_hostAOS_PosMass)
            goto Error;

        for ( size_t i = 0; i < 3; i++ ) {
            g_hostSOA_Pos[i] = (float *)valloc(sizeof(float) * g_N);
            if (!g_hostSOA_Pos[i])
                goto Error;

            g_hostSOA_Force[i] = (float *)valloc(sizeof(float) * g_N);
            if (!g_hostSOA_Force[i])
                goto Error;
        }
        g_hostSOA_Mass = (float *)valloc(sizeof(float) * g_N);
        if (!g_hostSOA_Mass)
            goto Error;

        g_hostAOS_Force = (float *)valloc(sizeof(float) * 3 * g_N);
        if (!g_hostAOS_Force)
            goto Error;

        g_hostAOS_Force_Golden = (float *)valloc(sizeof(float) * 3 * g_N);
        if (!g_hostAOS_Force_Golden)
            goto Error;

        g_hostAOS_VelInvMass = (float *)valloc(sizeof(float) * 4 * g_N);
        if (!g_hostAOS_VelInvMass)
            goto Error;

        g_hostSOA_InvMass = (float *)valloc(sizeof(float) * g_N);
        if (!g_hostSOA_InvMass)
            goto Error;
    }
    return 0;
Error:
    fprintf(stderr, "Failed to allocate required memory.\n");
    return 1;
}

static void usage(const char *argv0)
{
    printf( "Usage: nbody --bodies=N [--gpus=N] [--no-cpu] [--no-crosscheck] [--cycle-after=N] [--iterations=N]\n" );
    printf( "    --bodies is multiplied by 1024 (default is 16)\n" );
    printf( "    By default, the app checks results against a CPU implementation; \n" );
    printf( "    disable this behavior with --no-crosscheck.\n" );
    printf( "    The CPU implementation may be disabled with --no-cpu.\n" );
    printf( "    --no-cpu implies --no-crosscheck.\n\n" );
    printf( "    --iterations specifies a fixed number of iterations to execute\n" );
    printf( "    --cycle-after specifies the number of iterations before rotating\n" );
    printf( "                  to the next available algorithm\n" );
}

static inline int
processorCount(void)
{
#ifdef _OPENMP
    int k;
#  pragma omp parallel
    {
#  pragma omp master
        {
            k = omp_get_num_threads();
        }
    }
    return k;
#else
    return 1;
#endif
}

int
main( int argc, char *argv[] )
{
    hipError_t status;
    // kiloparticles
    int kParticles = 16, kMaxIterations = 0, kCycleAfter = 0;

    static const struct option cli_options[] = {
        { "bodies", required_argument, NULL, 'b' },
        { "gpus", required_argument, NULL, 'g' },
        { "no-cpu", no_argument, &g_bNoCPU, 1 },
        { "no-crosscheck", no_argument, &g_bCrossCheck, 0 },
        { "iterations", required_argument, NULL, 'i' },
        { "cycle-after", required_argument, NULL, 'c' },
        { "help", no_argument, NULL, 'h' },
        { NULL, 0, NULL, 0 }
    };

    libtime_init();

    status = hipGetDeviceCount( &g_numGPUs );
    if (status != hipSuccess)
        g_numGPUs = 0;

    while (1) {
        int option = getopt_long(argc, argv, "n:i:c:", cli_options, NULL);

        if (option == -1)
            break;

        switch (option) {
        case 'c':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--cycle-after'\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "ERROR: Requested cycle size less than 1\n");
                    return 1;
                }
                kCycleAfter = v;
            }
            break;
        case 'i':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--iterations'\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "ERROR: Requested number of iterations less than 1\n");
                    return 1;
                }
                kMaxIterations = v;
            }
            break;
        case 'b':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--bodies'\n");
                    return 1;
                }
                if (v < 1) {
                    printf("ERROR: Requested number of bodies less than 1");
                    return 1;
                }
                kParticles = v;
            }
            break;
        case 'g':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--gpus'\n");
                    return 1;
                }
                if (v < 1) {
                    if (g_numGPUs > 0)
                        fprintf(stderr, "Requested number of GPUs less than 1, disabling GPU algorithms.\n");
                    g_numGPUs = 0;
                    break;
                }
                if (v > g_numGPUs) {
                    fprintf(stderr, "Requested %d GPUs, but only have %d, using all available GPUs.\n",
                            v, g_numGPUs);
                    break;
                }
                g_numGPUs = v;
            }
            break;
        case 'h':
        case '?':
            usage(argv[0]);
            return 1;
        }
    }

    // for reproducible results for a given N
    srand(7);

    g_bCUDAPresent = g_numGPUs > 0;
    if ( g_bCUDAPresent ) {
        struct hipDeviceProp_t prop;
        CUDART_CHECK( hipGetDeviceProperties( &prop, 0 ) );
        g_bSM30Present = prop.major >= 3;
    }

    if ( g_bNoCPU && ! g_bCUDAPresent ) {
        fprintf(stderr, "ERROR: --no-cpu specified, but no CUDA present\n" );
        exit(1);
    }

    if ( g_numGPUs ) {
        g_GPUThreadPool = (worker_thread_t *)malloc(sizeof(worker_thread_t) * g_numGPUs);
        for (int i = 0; i < g_numGPUs; i++) {
            if (worker_create(&g_GPUThreadPool[i])) {
                fprintf( stderr, "Error initializing thread pool\n" );
                return 1;
            }
            if (worker_start(&g_GPUThreadPool[i])) {
                fprintf( stderr, "Error starting thread pool\n" );
                return 1;
            }
        }
        for ( int i = 0; i < g_numGPUs; i++ ) {
            struct gpuInit_struct initGPU = {i};
            worker_delegate(&g_GPUThreadPool[i], initializeGPU, &initGPU, 1);
            if ( hipSuccess != initGPU.status ) {
                fprintf( stderr, "Initializing GPU %d failed "
                    " with %d (%s)\n",
                    i,
                    initGPU.status,
                    hipGetErrorString( initGPU.status ) );
                return 1;
            }
        }
    }

    if ( g_bNoCPU ) {
        g_bCrossCheck = 0;
    }

    g_N = kParticles * 1024;

    printf( "Running simulation with %d particles, crosscheck %s, CPU %s, %d threads\n", (int) g_N,
        g_bCrossCheck ? "enabled" : "disabled",
        g_bNoCPU ? "disabled" : "enabled",
        processorCount() );

#if defined(HAVE_SIMD)
    g_maxAlgorithm = CPU_SIMD;
#else
    g_maxAlgorithm = CPU_SOA_tiled;
#endif
    g_Algorithm = g_bCUDAPresent ? GPU_AOS : CPU_SOA;
    if ( g_bCUDAPresent || g_bNoCPU ) {
        // max algorithm is different depending on whether SM 3.0 is present
        g_maxAlgorithm = g_bSM30Present ? GPU_AOS_tiled_const : multiGPU;
    }

    if (allocArrays() != 0)
        return 1;

    randomUnitBodies( g_hostAOS_PosMass, g_hostAOS_VelInvMass, g_N );
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Mass[i] = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    {
        int kIterations = 0;
        int bStop = 0;
        while ( ! bStop ) {
            float ms, err;

            if (ComputeGravitation(&ms, &err, g_Algorithm, g_bCrossCheck)) {
                fprintf( stderr, "Error computing timestep\n" );
                exit(1);
            }
            double interactionsPerSecond = (double) g_N*g_N*1000.0f / ms,
                   flops = interactionsPerSecond * (3 + 6 + 4 + 1 + 6) * 1e-3;
            if ( interactionsPerSecond > 1e9 ) {
                printf ( "\r%13s: %8.2f ms = %8.3fx10^9 interactions/s (%9.2lf GFLOPS)",
                    rgszAlgorithmNames[g_Algorithm],
                    ms,
                    interactionsPerSecond/1e9,
                    flops * 1e-6 );
            }
            else {
                printf ( "\r%13s: %8.2f ms = %8.3fx10^6 interactions/s (%9.2lf GFLOPS)",
                    rgszAlgorithmNames[g_Algorithm],
                    ms,
                    interactionsPerSecond/1e6,
                    flops * 1e-6 );
            }
            if (g_bCrossCheck)
                printf( " (Rel. error: %E)\n", err );
            else
                printf( "\n" );

            kIterations++;
            if (kMaxIterations) {
                int kIterationRatio = kCycleAfter * (g_maxAlgorithm + 1);
                if (!kIterationRatio)
                    kIterationRatio = 1;
                if (kIterations / kIterationRatio >= kMaxIterations) {
                    bStop = 1;
                }
            }
            if (kCycleAfter && kIterations % kCycleAfter == 0) {
                g_Algorithm = (enum nbodyAlgorithm_enum) (g_Algorithm+1);
                if ( g_Algorithm > g_maxAlgorithm ) {
                    g_Algorithm = g_bNoCPU ? GPU_AOS : CPU_AOS;
                }
            }
            if ( kbhit() ) {
                char c = getch();
                switch ( c ) {
                    case ' ':
                        g_Algorithm = (enum nbodyAlgorithm_enum) (g_Algorithm+1);
                        if ( g_Algorithm > g_maxAlgorithm ) {
                            g_Algorithm = g_bNoCPU ? GPU_AOS : CPU_AOS;
                        }
                        break;
                    case 'q':
                    case 'Q':
                        bStop = 1;
                        break;
                }

            }
        }
    }

    freeArrays();

    for ( int i = 0; i < g_numGPUs; i++ ) {
        struct gpuInit_struct initGPU = {i};
        worker_delegate(&g_GPUThreadPool[i], teardownGPU, &initGPU, 1);
        if ( hipSuccess != initGPU.status ) {
            fprintf( stderr, "GPU %d teardown failed "
                " with %d (%s)\n",
                i,
                initGPU.status,
                hipGetErrorString( initGPU.status ) );
            return 1;
        }
    }

    for (int i = 0; i < g_numGPUs; i++) {
        worker_delegate(&g_GPUThreadPool[i], NULL, NULL, 0);
        worker_join(&g_GPUThreadPool[i]);
        worker_destroy(&g_GPUThreadPool[i]);
    }

    return 0;
Error:
    if ( hipSuccess != status ) {
        printf( "CUDA Error: %s\n", hipGetErrorString( status ) );
    }
    return 1;
}

/* vim: set ts=4 sts=4 sw=4 et: */
