#include "hip/hip_runtime.h"
/*
 *
 * nbody.cu
 *
 * N-body example that illustrates gravitational simulation.
 * This is the type of computation that GPUs excel at:
 * parallelizable, with lots of FLOPS per unit of external
 * memory bandwidth required.
 *
 * Copyright (c) 2011-2012, Archaea Software, LLC.
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in
 *    the documentation and/or other materials provided with the
 *    distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 * FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 * COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 * INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 * ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <getopt.h>

#ifdef _WIN32
#include <conio.h>
#pragma comment (lib, "libtime.lib")
#pragma comment (lib, "libc11.lib")
#else
#include <malloc.h>
#endif

#include <math.h>

#include "libtime.h"

#include "chThread.h"
#include "chError.h"

#include "nbody.h"
#include "nbody_util.h"

#include "nbody_CPU_AOS.h"
#include "nbody_CPU_AOS_tiled.h"
#include "nbody_CPU_SOA.h"
#include "nbody_CPU_SOA_tiled.h"
#include "nbody_CPU_SIMD.h"

#ifndef NO_CUDA
#include "bodybodyInteraction.cuh"
#include "nbody_GPU_AOS.cuh"
#include "nbody_GPU_AOS_const.cuh"
//#include "nbody_GPU_AOS_tiled.cuh"
//#include "nbody_GPU_AOS_tiled_const.cuh"
//#include "nbody_GPU_SOA_tiled.cuh"
#include "nbody_GPU_Shuffle.cuh"
//#include "nbody_GPU_Atomic.cuh"
#endif

#ifdef HAVE_SIMD
#if defined(__HIPCC__)
// The platform-specific ISA macros aren't defined properly under CUDA, so we
// wouldn't get the right name. Let the algorithm itself declare its name.
extern const char *SIMD_ALGORITHM_NAME;
#elif defined(__ALTIVEC__)
#define SIMD_ALGORITHM_NAME "AltiVec"
#elif defined(__ARM_NEON__)
#define SIMD_ALGORITHM_NAME "NEON"
#elif defined(__AVX__)
#define SIMD_ALGORITHM_NAME "AVX"
#elif defined(__SSE__)
#define SIMD_ALGORITHM_NAME "SSE"
#else
#error "Define a name for this platform's SIMD."
#endif
#endif

#define DEFAULT_KPARTICLES 16

static const algorithm_def_t s_algorithms[] = {
	{ "CPU_SOA",             ALGORITHM_SOA,      { .soa = ComputeGravitation_SOA                 } },
	{ "CPU_SOA_tiled",       ALGORITHM_SOA,      { .soa = ComputeGravitation_SOA_tiled           } },
#ifdef HAVE_SIMD
	{ SIMD_ALGORITHM_NAME,   ALGORITHM_SOA,      { .soa = ComputeGravitation_SIMD                } },
#endif
	{ "CPU_AOS",             ALGORITHM_AOS,      { .aos = ComputeGravitation_AOS                 } },
	{ "CPU_AOS_tiled",       ALGORITHM_AOS,      { .aos = ComputeGravitation_AOS_tiled           } },
#ifndef NO_CUDA
	{ "GPU_AOS",             ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_AOS             } },
	{ "GPU_Shared",          ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_Shared          } },
	{ "GPU_Const",           ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_AOS_const       } },
//	{ "MultiGPU",            ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_multiGPU            } },
	{ "GPU_Shuffle",         ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_Shuffle         } },
//	{ "GPU_SOA_tiled",       ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_SOA_tiled       } },
//	{ "GPU_AOS_tiled",       ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_AOS_tiled       } },
//	{ "GPU_AOS_tiled_const", ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_AOS_tiled_const } },
//	{ "GPU_Atomic",          ALGORITHM_AOS_GPU,  { .aos = ComputeGravitation_GPU_Atomic          } },
#endif
	{ 0 },
};

static int maxAlgorithmIdx(void)
{
    static int idx = -1;
    if (idx != -1)
        return idx;
    for (idx = 0; s_algorithms[idx].name; idx++);
    idx--;
    return idx;
}

static float
relError( float a, float b )
{
    if ( a == b ) return 0.0f;
    return fabsf(a-b)/b;
}

static int g_bCUDAPresent;

afloat *g_hostAOS_PosMass;
afloat *g_hostAOS_VelInvMass;
afloat *g_hostAOS_Force;

#ifndef NO_CUDA
static afloat *g_dptrAOS_PosMass;
static afloat *g_dptrAOS_Force;
#endif

// Buffer to hold the golden version of the forces, used for comparison
// Along with timing results, we report the maximum relative error with
// respect to this array.
static afloat *g_hostAOS_Force_Golden;

afloat *g_hostSOA_Pos[3];
afloat *g_hostSOA_Force[3];
afloat *g_hostSOA_Mass;
afloat *g_hostSOA_InvMass;

static size_t g_N;

static afloat g_softening = 0.1f;
static afloat g_damping = 0.995f;
static afloat g_dt = 0.016f;

static void
integrateGravitation_AOS( afloat *ppos, afloat *pvel, afloat *pforce, afloat dt, afloat damping, size_t N )
{
    for ( size_t i = 0; i < N; i++ ) {
        const int index = 4*i;

        ufloat pos[3], vel[3], force[3];
        pos[0] = ppos[index+0];
        pos[1] = ppos[index+1];
        pos[2] = ppos[index+2];
        ufloat invMass = pvel[index+3];

        vel[0] = pvel[index+0];
        vel[1] = pvel[index+1];
        vel[2] = pvel[index+2];

        force[0] = pforce[index+0];
        force[1] = pforce[index+1];
        force[2] = pforce[index+2];

        // acceleration = force / mass;
        // new velocity = old velocity + acceleration * deltaTime
        vel[0] += (force[0] * invMass) * dt;
        vel[1] += (force[1] * invMass) * dt;
        vel[2] += (force[2] * invMass) * dt;

        vel[0] *= damping;
        vel[1] *= damping;
        vel[2] *= damping;

        // new position = old position + velocity * deltaTime
        pos[0] += vel[0] * dt;
        pos[1] += vel[1] * dt;
        pos[2] += vel[2] * dt;

        ppos[index+0] = pos[0];
        ppos[index+1] = pos[1];
        ppos[index+2] = pos[2];

        pvel[index+0] = vel[0];
        pvel[index+1] = vel[1];
        pvel[index+2] = vel[2];
    }
}

static int g_bCrossCheck = 1;
static int g_bNoCPU = 0;

static int
ComputeGravitation(
    float *ms,
    float *maxRelError,
    const algorithm_def_t *algorithm,
    int bCrossCheck )
{
#ifndef NO_CUDA
    hipError_t status;
#endif
    int bSOA = 0;

    if (g_bNoCPU && algorithm->type != ALGORITHM_AOS_GPU)
        return 1;

    // AOS -> SOA data structures in case we are measuring SOA performance
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Pos[0][i]  = g_hostAOS_PosMass[4*i+0];
        g_hostSOA_Pos[1][i]  = g_hostAOS_PosMass[4*i+1];
        g_hostSOA_Pos[2][i]  = g_hostAOS_PosMass[4*i+2];
        g_hostSOA_Mass[i]    = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    if ( bCrossCheck && algorithm != &s_algorithms[0] ) {
        ComputeGravitation_SOA(
                        g_hostSOA_Force,
                        g_hostSOA_Pos,
                        g_hostSOA_Mass,
                        g_softening*g_softening,
                        g_N );
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force_Golden[4*i+0] = g_hostSOA_Force[0][i];
            g_hostAOS_Force_Golden[4*i+1] = g_hostSOA_Force[1][i];
            g_hostAOS_Force_Golden[4*i+2] = g_hostSOA_Force[2][i];
        }
    }

    /* Reset the force values so we know the function tested did work. */
    memset(g_hostAOS_Force,    0, g_N * sizeof(afloat) * 4);
    memset(g_hostSOA_Force[0], 0, g_N * sizeof(afloat));
    memset(g_hostSOA_Force[1], 0, g_N * sizeof(afloat));
    memset(g_hostSOA_Force[2], 0, g_N * sizeof(afloat));

    switch ( algorithm->type ) {
        case ALGORITHM_SOA:
            *ms = algorithm->soa(
                g_hostSOA_Force,
                g_hostSOA_Pos,
                g_hostSOA_Mass,
                g_softening*g_softening,
                g_N );
            bSOA = 1;
            break;
        case ALGORITHM_AOS:
            *ms = algorithm->aos(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
#ifndef NO_CUDA
        case ALGORITHM_AOS_GPU:
            CUDART_CHECK( hipMemcpyAsync(
                g_dptrAOS_PosMass,
                g_hostAOS_PosMass,
                4*g_N*sizeof(afloat),
                hipMemcpyHostToDevice ) );
            CUDART_CHECK( hipMemset( g_dptrAOS_Force, 0, 4*g_N*sizeof(afloat) ) );
            *ms = algorithm->aos(
                g_dptrAOS_Force,
                g_dptrAOS_PosMass,
                g_softening*g_softening,
                g_N );
            CUDART_CHECK( hipMemcpy( g_hostAOS_Force, g_dptrAOS_Force, 4*g_N*sizeof(afloat), hipMemcpyDeviceToHost ) );
            break;
#endif
#if 0
            // HOSTAOS?
        case multiGPU:
            memset( g_hostAOS_Force, 0, 4*g_N*sizeof(afloat) );
            *ms = ComputeGravitation_multiGPU(
                g_hostAOS_Force,
                g_hostAOS_PosMass,
                g_softening*g_softening,
                g_N );
            break;
#endif
        default:
            fprintf(stderr, "Unrecognized algorithm index: %d\n", algorithm->type);
            abort();
    }

    if ( *ms < __FLT_EPSILON__ )
        return 1;

    // SOA -> AOS
    if ( bSOA ) {
        for ( size_t i = 0; i < g_N; i++ ) {
            g_hostAOS_Force[4*i+0] = g_hostSOA_Force[0][i];
            g_hostAOS_Force[4*i+1] = g_hostSOA_Force[1][i];
            g_hostAOS_Force[4*i+2] = g_hostSOA_Force[2][i];
        }
    }

    *maxRelError = 0.0f;
    if ( bCrossCheck && algorithm != &s_algorithms[0] ) {
        float max = 0.0f;
        for ( size_t i = 0; i < 4*g_N; i++ ) {
            if ((i + 1) % 4 == 0)
                continue;
            float err = relError( g_hostAOS_Force[i], g_hostAOS_Force_Golden[i] );
            if ( err > max ) {
                max = err;
            }
        }
        *maxRelError = max;
    }

    integrateGravitation_AOS(
        g_hostAOS_PosMass,
        g_hostAOS_VelInvMass,
        g_hostAOS_Force,
        g_dt,
        g_damping,
        g_N );
    return 0;
#ifndef NO_CUDA
Error:
    return 1;
#endif
}

static worker_thread_t *g_GPUThreadPool;
int g_maxGPUs;
int g_numGPUs;

struct gpuInit_struct
{
    int iGPU;

    hipError_t status;
};

static int initializeGPU( void *_p )
{
    hipError_t status;

    struct gpuInit_struct *p = (struct gpuInit_struct *) _p;
    CUDART_CHECK( hipSetDevice( p->iGPU ) );
    CUDART_CHECK( hipFree(0) );
Error:
    p->status = status;
    return 0;
}

static int teardownGPU( void *_p )
{
    hipError_t status;

    struct gpuInit_struct *p = (struct gpuInit_struct *) _p;
    CUDART_CHECK( hipSetDevice( p->iGPU ) );
    CUDART_CHECK( hipDeviceReset() );
Error:
    p->status = status;
    return 0;
}

static int freeArrays(void)
{
#ifndef NO_CUDA
    hipError_t status;

    if ( g_bCUDAPresent ) {
        CUDART_CHECK( hipDeviceSynchronize() );
        CUDART_CHECK( hipHostFree( g_hostAOS_PosMass ) );
        for ( size_t i = 0; i < 3; i++ ) {
            CUDART_CHECK( hipHostFree( g_hostSOA_Pos[i] ) );
            CUDART_CHECK( hipHostFree( g_hostSOA_Force[i] ) );
        }
        CUDART_CHECK( hipHostFree( g_hostAOS_Force ) );
        CUDART_CHECK( hipHostFree( g_hostAOS_Force_Golden ) );
        CUDART_CHECK( hipHostFree( g_hostAOS_VelInvMass ) );
        CUDART_CHECK( hipHostFree( g_hostSOA_Mass ) );
        CUDART_CHECK( hipHostFree( g_hostSOA_InvMass ) );

        CUDART_CHECK( hipFree( g_dptrAOS_PosMass ) );
        CUDART_CHECK( hipFree( g_dptrAOS_Force ) );
    } else
#endif
    {
        alignedFree(g_hostAOS_PosMass);
        for ( size_t i = 0; i < 3; i++ ) {
            alignedFree(g_hostSOA_Pos[i]);
            alignedFree(g_hostSOA_Force[i]);
        }
        alignedFree(g_hostAOS_Force);
        alignedFree(g_hostAOS_Force_Golden);
        alignedFree(g_hostAOS_VelInvMass);
        alignedFree(g_hostSOA_Mass);
        alignedFree(g_hostSOA_InvMass);
    }
    return 0;
#ifndef NO_CUDA
Error:
    fprintf(stderr, "Failed to clean up memory.\n");
    return 1;
#endif
}

static int allocArrays(void)
{
#ifndef NO_CUDA
    hipError_t status;

    if ( g_bCUDAPresent ) {
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_PosMass, 4*g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );
        for ( size_t i = 0; i < 3; i++ ) {
            CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Pos[i], g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );
            CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Force[i], g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );
        }
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force, 4*g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_Force_Golden, 4*g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostAOS_VelInvMass, 4*g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_Mass, g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );
        CUDART_CHECK( hipHostAlloc( (void **) &g_hostSOA_InvMass, g_N*sizeof(afloat), hipHostMallocPortable|hipHostMallocMapped ) );

        CUDART_CHECK( hipMalloc( &g_dptrAOS_PosMass, 4*g_N*sizeof(afloat) ) );
        CUDART_CHECK( hipMalloc( (void **) &g_dptrAOS_Force, 4*g_N*sizeof(afloat) ) );
    } else
#endif
    {
        g_hostAOS_PosMass = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * 4 * g_N);
        if (!g_hostAOS_PosMass)
            goto Error;

        for ( size_t i = 0; i < 3; i++ ) {
            g_hostSOA_Pos[i] = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * g_N);
            if (!g_hostSOA_Pos[i])
                goto Error;

            g_hostSOA_Force[i] = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * g_N);
            if (!g_hostSOA_Force[i])
                goto Error;
        }
        g_hostSOA_Mass = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * g_N);
        if (!g_hostSOA_Mass)
            goto Error;

        g_hostAOS_Force = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * 4 * g_N);
        if (!g_hostAOS_Force)
            goto Error;

        g_hostAOS_Force_Golden = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * 4 * g_N);
        if (!g_hostAOS_Force_Golden)
            goto Error;

        g_hostAOS_VelInvMass = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * 4 * g_N);
        if (!g_hostAOS_VelInvMass)
            goto Error;

        g_hostSOA_InvMass = (afloat *)alignedAlloc(NBODY_ALIGNMENT, sizeof(afloat) * g_N);
        if (!g_hostSOA_InvMass)
            goto Error;
    }
    return 0;
Error:
    fprintf(stderr, "Failed to allocate required memory.\n");
    return 1;
}

static void print_algorithms(void)
{
    int idx = 0;
    fprintf(stderr, "Algorithms available in this build:\n\n");
    for (idx = 0; s_algorithms[idx].name; idx++) {
        fprintf(stdout, "   %d - %s\n", idx, s_algorithms[idx].name);
    }
#ifdef NO_CUDA
    fprintf(stderr, "\nThis build does not have CUDA support enabled. All GPU algorithms are unavailable.\n");
#endif
    fprintf(stderr, "\n");
}

static void print_usage(const char *argv0)
{
	fprintf(stderr, "Usage: nbody [arguments]\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "Available arguments:\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "	--bodies=<N> | -n <N>\n");
	fprintf(stderr, "		Specifies the number of random bodies to use in the simulation. The\n");
	fprintf(stderr, "		number is multiplied by 1024. [default: %d]\n", DEFAULT_KPARTICLES);
	fprintf(stderr, "\n");
	fprintf(stderr, "	--gpus=<N> | -g <N>\n");
	fprintf(stderr, "		Specifies the number of GPUs to use for the GPU-based algorithms.\n");
	fprintf(stderr, "		[default: number of available GPUs]\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "	--no-cpu\n");
	fprintf(stderr, "		Disables all CPU-based simulations (including crosscheck). Only makes\n");
	fprintf(stderr, "		sense if GPU-based algorithms are available.\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "	--no-crosscheck\n");
	fprintf(stderr, "		Disables cross-validation of results against a CPU implementation.\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "	--iterations=<N> | -i <N>\n");
	fprintf(stderr, "		Specifies the number of iterations through the algorithm list.\n");
	fprintf(stderr, "		[default: loop forever]\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "	--cycle-after=<N> | -c <N>\n");
	fprintf(stderr, "		Specifies the number of simulations steps to execute before cycling to\n");
	fprintf(stderr, "		the next available algorithm. [default: none, don't cycle]\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "	--list\n");
	fprintf(stderr, "		Lists the available simulation algorithms.\n");
	fprintf(stderr, "\n");
	fprintf(stderr, "	--algorithm=<name or index> | -a <name or index>\n");
	fprintf(stderr, "		Specifies a specific algorithm name or index to start with. See --list\n");
	fprintf(stderr, "		for the list of available algorithms for this argument.\n");
	fprintf(stderr, "		[default: %s]\n", s_algorithms[0].name);
	fprintf(stderr, "\n");
	fprintf(stderr, "	--help\n");
	fprintf(stderr, "		Prints this help text.\n");
}

int main(int argc, char **argv)
{
    hipError_t status;

    // kiloparticles
    int kParticles = DEFAULT_KPARTICLES, maxIterations = 0, cycleAfter = 0;
    int idxFirstAlgorithm = 0;

    static const struct option cli_options[] = {
        { "bodies", required_argument, NULL, 'n' },
        { "gpus", required_argument, NULL, 'g' },
        { "no-cpu", no_argument, &g_bNoCPU, 1 },
        { "no-crosscheck", no_argument, &g_bCrossCheck, 0 },
        { "iterations", required_argument, NULL, 'i' },
        { "cycle-after", required_argument, NULL, 'c' },
        { "list", no_argument, NULL, 'l' },
        { "algorithm", required_argument, NULL, 'a' },
        { "help", no_argument, NULL, 'h' },
        { NULL, 0, NULL, 0 }
    };

    status = hipGetDeviceCount( &g_maxGPUs );
    if (status != hipSuccess)
        g_numGPUs = 0;
    else
        g_numGPUs = g_maxGPUs;

    while (1) {
        int option = getopt_long(argc, argv, "n:i:c:g:la:h", cli_options, NULL);

        if (option == -1)
            break;

        switch (option) {
        case 'c':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--cycle-after'\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "ERROR: Requested cycle size less than 1\n");
                    return 1;
                }
                cycleAfter = v;
            }
            break;
        case 'i':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--iterations'\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "ERROR: Requested number of iterations less than 1\n");
                    return 1;
                }
                maxIterations = v;
            }
            break;
        case 'n':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--bodies'\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "ERROR: Requested number of bodies less than 1\n");
                    return 1;
                }
                kParticles = v;
            }
            break;
        case 'g':
            {
                int v;
                if (sscanf(optarg, "%d", &v) != 1) {
                    fprintf(stderr, "ERROR: Couldn't parse integer argument for '--gpus'\n");
                    return 1;
                }
                if (g_maxGPUs < 1) {
                    if (v == 0) {
                        g_numGPUs = 0;
                        break;
                    }
                    fprintf(stderr, "ERROR: No GPUs available, cannot handle '--gpus' argument.\n");
                    return 1;
                }
                if (v < 1) {
                    fprintf(stderr, "Requested number of GPUs less than 1, disabling GPU algorithms.\n");
                    g_numGPUs = 0;
                    break;
                }
                if (v > g_maxGPUs) {
                    fprintf(stderr, "Requested %d GPUs, but only have %d, using all available GPUs.\n",
                            v, g_maxGPUs);
                    v = g_maxGPUs;
                }
                g_numGPUs = v;
            }
            break;
        case 'a':
            {
                const char *type;
                int v;

                idxFirstAlgorithm = -1;

                // First try to parse as an integer argument
                if (sscanf(optarg, "%d", &v) == 1) {
                    type = "index";
                    if (v >= 0 && v <= maxAlgorithmIdx()) {
                        idxFirstAlgorithm = v;
                    }
                } else {
                    type = "name";
                    // Alternatively, check if it's a valid algorithm name
                    for (v = 0; s_algorithms[v].name; v++) {
                        int n;
                        n = strcasecmp(optarg, s_algorithms[v].name);
                        if (n == 0) {
                            // Exact match
                            idxFirstAlgorithm = v;
                            break;
                        }
                        n = strncasecmp(optarg, s_algorithms[v].name, strlen(optarg));
                        if (n == 0) {
                            // Partial match, keep searching in case there's an
                            // exact match
                            idxFirstAlgorithm = v;
                            continue;
                        }
                    }
                }

                if (idxFirstAlgorithm == -1) {
                    fprintf(stderr, "Invalid algorithm %s '%s'\n\n", type, optarg);
                    print_algorithms();
                    return 1;
                }
            }
            break;
        case 'l':
            print_algorithms();
            return 1;
        case 'h':
        case '?':
            print_usage(argv[0]);
            return 1;
        }
    }

    libtime_init();

    // for reproducible results for a given N
    seedRandom(7);

    g_bCUDAPresent = g_numGPUs > 0;
    if ( g_bCUDAPresent ) {
        struct hipDeviceProp_t prop;
        CUDART_CHECK( hipGetDeviceProperties( &prop, 0 ) );
    }

    if ( g_bNoCPU && ! g_bCUDAPresent ) {
        fprintf(stderr, "ERROR: --no-cpu specified, but no CUDA present\n" );
        exit(1);
    }

    if ( g_numGPUs ) {
        g_GPUThreadPool = (worker_thread_t *)malloc(sizeof(worker_thread_t) * g_numGPUs);
        for (int i = 0; i < g_numGPUs; i++) {
            if (worker_create(&g_GPUThreadPool[i])) {
                fprintf(stderr, "Error initializing thread pool\n");
                return 1;
            }
            if (worker_start(&g_GPUThreadPool[i])) {
                fprintf(stderr, "Error starting thread pool\n");
                return 1;
            }
        }
        for ( int i = 0; i < g_numGPUs; i++ ) {
            struct gpuInit_struct initGPU = {i};
            worker_delegate(&g_GPUThreadPool[i], initializeGPU, &initGPU, 1);
            if ( hipSuccess != initGPU.status ) {
                fprintf(stderr, "Initializing GPU %d failed "
                    " with %d (%s)\n",
                    i,
                    initGPU.status,
                    hipGetErrorString( initGPU.status ));
                return 1;
            }
        }
    }

    if ( g_bNoCPU ) {
        g_bCrossCheck = 0;
    }

    g_N = kParticles * 1024;

    fprintf(stderr, "Running simulation with %d particles, crosscheck %s, CPU %s, %d threads\n", (int) g_N,
        g_bCrossCheck ? "enabled" : "disabled",
        g_bNoCPU ? "disabled" : "enabled",
        processorCount() );

    if (allocArrays() != 0)
        return 1;

    randomUnitBodies( g_hostAOS_PosMass, g_hostAOS_VelInvMass, g_N );
    for ( size_t i = 0; i < g_N; i++ ) {
        g_hostSOA_Mass[i] = g_hostAOS_PosMass[4*i+3];
        g_hostSOA_InvMass[i] = 1.0f / g_hostSOA_Mass[i];
    }

    {
        int algorithm_idx = idxFirstAlgorithm;
        int steps = 0, iterations = 0;
        int bStop = 0;
        while ( ! bStop ) {
            float ms, err;
            const algorithm_def_t *algorithm = &s_algorithms[algorithm_idx];

            if (!ComputeGravitation(&ms, &err, algorithm, g_bCrossCheck))
            {
                double interactionsPerSecond = (double) g_N*g_N*1000.0f / ms,
                       flops = interactionsPerSecond * (3 + 6 + 4 + 1 + 6) * 1e-3;

                if ( interactionsPerSecond > 1e9 )
                {
                    fprintf(stdout, "\r%13s: %8.2f ms = %8.3fx10^9 interactions/s (%9.2lf GFLOPS)",
                        algorithm->name,
                        ms,
                        interactionsPerSecond/1e9,
                        flops * 1e-6 );
                }
                else
                {
                    fprintf(stdout, "\r%13s: %8.2f ms = %8.3fx10^6 interactions/s (%9.2lf GFLOPS)",
                        algorithm->name,
                        ms,
                        interactionsPerSecond/1e6,
                        flops * 1e-6 );
                }
                if (g_bCrossCheck)
                    fprintf(stdout, " (Rel. error: %E)\n", err );
                else
                    fprintf(stdout, "\n" );
            } else {
                goto next_algorithm;
            }

            steps++;
            if (cycleAfter && steps % cycleAfter == 0) {
next_algorithm:
                algorithm_idx++;
                if ( !s_algorithms[algorithm_idx].name ) {
                    algorithm_idx = 0;
                    iterations++;
                }
            } else if (!cycleAfter) {
                iterations++;
            }
            if (maxIterations && iterations >= maxIterations) {
                bStop = 1;
            }
            if ( kbhit() ) {
                char c = getch();
                switch ( c ) {
                    case ' ':
                        algorithm_idx++;
                        if ( !s_algorithms[algorithm_idx].name ) {
                            algorithm_idx = 0;
                            iterations++;
                        }
                        break;
                    case 'q':
                    case 'Q':
                        bStop = 1;
                        break;
                }

            }
        }
    }

    freeArrays();

    for ( int i = 0; i < g_numGPUs; i++ ) {
        struct gpuInit_struct initGPU = {i};
        worker_delegate(&g_GPUThreadPool[i], teardownGPU, &initGPU, 1);
        if ( hipSuccess != initGPU.status ) {
            fprintf(stderr, "GPU %d teardown failed "
                " with %d (%s)\n",
                i,
                initGPU.status,
                hipGetErrorString( initGPU.status ) );
            return 1;
        }
    }

    for (int i = 0; i < g_numGPUs; i++) {
        worker_delegate(&g_GPUThreadPool[i], NULL, NULL, 0);
        worker_join(&g_GPUThreadPool[i]);
        worker_destroy(&g_GPUThreadPool[i]);
    }

    return 0;
Error:
    if ( hipSuccess != status ) {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString( status ) );
    }
    return 1;
}

/* vim: set ts=4 sts=4 sw=4 et: */
